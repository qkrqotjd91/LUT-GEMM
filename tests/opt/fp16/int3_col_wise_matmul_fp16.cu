#include "hip/hip_runtime.h"
#include "tests.h"


// void t_matmul_useCublas(float* output, lutGEMM::nQWeight_fp16 nqW, float* input, int n) {
//     lutGEMM::kernel::cublas_gemm_ex(nqW.getDequantiedWeight(), input, output, nqW.mSize, nqW.kSize, n);
// }


#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>



template <typename T, typename S>
inline hipblasStatus_t cublas_gemm_ex(T *A,  T *B,  S *C,
                                    int m, int n, int k);
                                    
template<int M, int N, int K, int NUM_BITS, int A_GROUP_SIZE=K>
class int3_col_wise_matmul_fp16{
public:
    static const int num_groups = K/A_GROUP_SIZE;
    float     qW[K   ][NUM_BITS][N]; // (-1, 1) 
    uint32_t  bW[K/32][NUM_BITS][N]; // bit packed
    float     alpha[num_groups][NUM_BITS][N];
    float    q_bias[num_groups][N];

    float   weight[K][N];           // float weight
    float    input[M][K];
    float   output[M][N];

    int K_new = K * 3 / 32; // 3bit weights are packed into int32
    int   weight_int3[K * 3 / 32][N];
    float scale[N];
    float bias[N];

    int*    d_weight_int3;
    __half* d_scale;
    __half* d_bias;
    __half* d_gptq_output;


    __half* d_weight_fp16;
    __half*  d_input;

    __half* d_cu_output;
    __half* d_nq_output;

    lutGEMM::nQWeight_fp16 nqW;

    double run(bool run_cublas=true, bool run_lutgemm=false, bool run_gptq=false, int iter=16){
        alloc_cuda();
        makeRandomInput();
        makeRandomWeight();
        makeRandomWeight_int3();
        makeRandomAlpha();
        //dequantizeFrom_qW();
        copy_cpuToCuda();

        nqW.parsing((uint32_t*)bW, (float*)alpha, K, N, NUM_BITS, false, num_groups, (float*)q_bias);
        hipDeviceSynchronize();

        //double meanError = checkErr();
        double meanError = 0;
        hipDeviceSynchronize();

        if(run_cublas) cublas_latency(M, N, K, d_input, d_weight_fp16, d_cu_output, iter);
        if(run_lutgemm) lutgemm_latency(nqW, M, N, K, d_input, d_weight_fp16, d_cu_output, iter);
        //if(run_gptq) gptq_latency(M, N, K_new, d_scale, d_bias, d_input, d_weight_int3, d_gptq_output, iter);
        if(run_gptq) gptq_faster_latency(M, N, K_new, d_scale, d_bias, d_input, d_weight_int3, d_gptq_output, iter);

        free_cuda();
        return meanError;
    }

    void gptq_latency(int m, int n, int k, __half* scale, __half* bias, __half* A, int *B, __half *C, int iter=64){
        timer tm;

        lutGEMM::matmul_gptq(m, n, k, (void*)scale, (void*)bias,
                        (void*)A, (void*)B, (void*)C);
        hipDeviceSynchronize();

        for(int i=0;i<iter;i++){
            tm.start();
            lutGEMM::matmul_gptq(m, n, k, (void*)scale, (void*)bias,
                        (void*)A, (void*)B, (void*)C);
            hipDeviceSynchronize();
            tm.end();
        }
        printf("latency min : %.5fms, max : %.5fms, avg:%.5f\n", tm.min(), tm.max(), tm.mean());
    }

    void gptq_faster_latency(int m, int n, int k, __half* scale, __half* bias, __half* A, int *B, __half *C, int iter=64){
        timer tm;

        lutGEMM::matmul_gptq_faster(m, n, k, (void*)scale, (void*)bias,
                        (void*)A, (void*)B, (void*)C);
        hipDeviceSynchronize();

        for(int i=0;i<iter;i++){
            tm.start();
            lutGEMM::matmul_gptq_faster(m, n, k, (void*)scale, (void*)bias,
                        (void*)A, (void*)B, (void*)C);
            hipDeviceSynchronize();
            tm.end();
        }
        printf("latency min : %.5fms, max : %.5fms, avg:%.5f\n", tm.min(), tm.max(), tm.mean());
    }

    void lutgemm_latency(lutGEMM::nQWeight_fp16 &nqW, int m, int n, int k, __half* A, __half *B, __half *C, int iter=64){
        timer tm;

        lutGEMM::matmul((void*)C, (void*)A, nqW, m);
        hipDeviceSynchronize();

        for(int i=0;i<iter;i++){
            tm.start();
            lutGEMM::matmul((void*)C, (void*)A, nqW, m);
            hipDeviceSynchronize();
            tm.end();
        }
        printf("latency min : %.5fms, max : %.5fms, avg:%.5f\n", tm.min(), tm.max(), tm.mean());
    }

    void cublas_latency(int m, int n, int k, __half* A, __half *B, __half *C, int iter=64){
        timer tm;
        float th = 0;
        cublas_gemm_ex(A, B, C,
                            m, n, k);
        hipDeviceSynchronize();
        for (int i = 0; i < iter; ++i) {
            tm.start();
            hipblasStatus_t success;
            success = cublas_gemm_ex(A, B, C,
                                    m, n, k);
            hipDeviceSynchronize();
            tm.end();

        }
            printf("latency min : %.5fms, max : %.5fms, avg:%.5f\n", tm.min(), tm.max(), tm.mean());

    }


    double checkErr(){
        cublas_gemm_ex(d_input, d_weight_fp16, d_cu_output, M, N, K);
        hipMemset(d_nq_output, 0, sizeof(float) * M * N);
        lutGEMM::matmul(d_nq_output, d_input, nqW, M);
        hipDeviceSynchronize();
        return checkOutputMeanError(d_cu_output, d_nq_output);
    }

    double checkOutputMeanError(__half *o1, __half *o2){
        double err=0;
        for(int m=0;m<M;m++){
            for(int n=0;n<N;n++){
                err += std::abs(float(o1[m*N + n]) - float(o2[m*N + n]));
                // if(n<100) printf("%f %f\n", float(o1[m*N + n]), float(o2[m*N + n]));
            }
        }
        return err/M/N;
    }

    void matmul_cpu(){
        for(int m=0;m<M;m++){
            for(int n=0;n<N;n++){
                output[m][n] = 0;
                for(int k=0;k<K;k++){
                    output[m][n] += input[m][k] * weight[k][n];
                }
            }
        }
    }

    void makeRandomInput(){
        for(int m=0;m<M;m++)
            for(int k=0;k<K;k++)
                input[m][k] = rand_fp32(); // (-1.0, 1.0) / 2^b
    }

    void makeRandomAlpha(){
        for(int g=0;g<num_groups;g++)
            for(int n=0;n<N;n++){
                q_bias[g][n] = rand_fp32()/(1<< NUM_BITS);
                for(int b=0;b<NUM_BITS;b++)
                    alpha[g][b][n] = rand_fp32()/(1<<b); // (-1.0, 1.0) / 2^b
            }
    }

    void makeRandomWeight(){
        for(int n=0;n<N;n++){
            for(int b=0;b<NUM_BITS;b++){
                for(int k=0;k<K;k+=32){  //32 단위
                    uint32_t s=0;
                    for(int t=0;t<32;t++){
                        if(rand_bool()){
                                s |= 1<<t;
                                qW[k + t][b][n] = +1;
                        } else  qW[k + t][b][n] = -1;
                    }
                    bW[k/32][b][n] = s;
                }
            }
        }
    }

    void makeRandomWeight_int3(){
        for(int n=0;n<N;n++){
            for(int k=0;k<K_new;k++){
                weight_int3[k][n] = rand();
            }
        }
    }

    void makeRandomScale(){
        for(int n=0;n<N;n++)
            scale[n] = rand_fp32();
    }

    void makeRandomBias(){
        for(int n=0;n<N;n++)
            bias[n] = rand_fp32();
    }

    void dequantizeFrom_qW(){
        for(int n=0;n<N;n++){
            for(int k=0;k<K;k++){  //32 단위
                weight[k][n] = q_bias[k/A_GROUP_SIZE][n];
                for(int b=0;b<NUM_BITS;b++){
                    weight[k][n] += alpha[k/A_GROUP_SIZE][b][n] * qW[k][b][n]; 
                }
            }
        }        
    }    

    void alloc_cuda(){
        hipMallocManaged(&d_input    , sizeof(float) * M * K);   
        hipMallocManaged(&d_weight_fp16, sizeof(float) * K * N);   

        hipMallocManaged(&d_cu_output, sizeof(float) * M * N);       
        hipMallocManaged(&d_nq_output, sizeof(float) * M * N);

        hipMallocManaged(&d_weight_int3, sizeof(int) * K_new * N);   
        hipMallocManaged(&d_scale, sizeof(float) * N);   
        hipMallocManaged(&d_bias, sizeof(float) * N);   
        hipMallocManaged(&d_gptq_output, sizeof(float) * M * N);

    }
    
    void free_cuda(){
        hipFree(d_input);
        hipFree(d_weight_fp16);
        hipFree(d_cu_output);
        hipFree(d_nq_output);

        hipFree(d_weight_int3);
        hipFree(d_scale);
        hipFree(d_bias);
        hipFree(d_gptq_output);
    }
    void copy_cpuToCuda(){
        fhCpy(d_input , (float*)input  ,M * K);
        fhCpy(d_weight_fp16, (float*)weight ,K * N);

        hipMemcpy(d_weight_int3, (int*)weight_int3,
            K_new * N, hipMemcpyHostToDevice);
        fhCpy(d_scale, (float*)scale , N);
        fhCpy(d_bias, (float*)bias , N);

        hipDeviceSynchronize();
    }

    void hfCpy(float* a, __half* b, int size){
       for(int i=0;i<size;i++) a[i] = float(b[i]);
    }
    void fhCpy(__half* a, float* b, int size){
       for(int i=0;i<size;i++) a[i] = __float2half(b[i]);
    }

};

const int H = 7168;
TEST(int3_col_wise_matmul_fp16, layer_175b){
    double total_error = 0;
    int e_cnt = 0;

    { auto t = std::make_shared<int3_col_wise_matmul_fp16<1, H*4, H, 3, 128>>(); total_error += t->run(true, true, true); e_cnt++; }
    printf("----------------------------------------------------------------\n");
    printf("Warm up done.\n");
    printf("----------------------------------------------------------------\n");
    printf("M = 1, N = %d, K = %d\n", 4*H, H);
    printf("cuBLAS [FP16, FP16, FP16]\t");
    { auto t = std::make_shared<int3_col_wise_matmul_fp16<1, H*4, H, 3, 128>>(); total_error += t->run(true, false, false); e_cnt++; }
    printf("OPTQ [INT3, FP16, FP16]\t\t");
    { auto t = std::make_shared<int3_col_wise_matmul_fp16<1, H*4, H, 4, 128>>(); total_error += t->run(false, false, true); e_cnt++; }  

    printf("LUT-GEMM [INT8, FP16, FP16]\t");
    { auto t = std::make_shared<int3_col_wise_matmul_fp16<1, H*4, H, 8, 128>>(); total_error += t->run(false, true, false); e_cnt++; } 
    
    printf("LUT-GEMM [INT4, FP16, FP16]\t");
    { auto t = std::make_shared<int3_col_wise_matmul_fp16<1, H*4, H, 4, 128>>(); total_error += t->run(false, true, false); e_cnt++; }  
    printf("LUT-GEMM [INT3, FP16, FP16]\t");
    { auto t = std::make_shared<int3_col_wise_matmul_fp16<1, H*4, H, 3, 128>>(); total_error += t->run(false, true, false); e_cnt++; }
}




template <typename T, typename S>
inline hipblasStatus_t cublas_gemm_ex(T *A,  T *B,  S *C,
                                    int m, int n, int k) {
    static S alpha = 1;
    static S beta  = 0;
    static hipblasHandle_t handle = nullptr;
    if(handle == nullptr) hipblasCreate(&handle);
    
    hipDataType AType, BType, CType;
    hipblasComputeType_t  ComputeType;
    if (std::is_same<T, float>::value) {
        AType = BType = CType = HIP_R_32F;
        ComputeType = HIPBLAS_COMPUTE_32F_FAST_TF32;
    } else if (std::is_same<T, __half>::value) {
        AType = BType = CType = HIP_R_16F;
        ComputeType = HIPBLAS_COMPUTE_16F;
    } else if (std::is_same<T, int8_t>::value) {
        AType = BType = HIP_R_8I;
        CType = HIP_R_32I;
        ComputeType = HIPBLAS_COMPUTE_32I;
    } else {
        return HIPBLAS_STATUS_NOT_SUPPORTED;
    }
    return hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                          n, m, k, 
                          &alpha,
                          B, BType, n,
                          A, AType, k,
                          &beta,
                          C, CType, n,
                          ComputeType,
                          HIPBLAS_GEMM_DEFAULT);
}