#include "hip/hip_runtime.h"
#include "../include/kernels.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

namespace lutGEMM{

#include "../src/cuda/kernels/hipblas.h"
#include "../src/cuda/kernels/mv_fp16.hpp"
#include "../src/cuda/kernels/mv_fp16_bias.hpp"
#include "../src/cuda/kernels/gptq_fp16_bias.hpp"
#include "../src/cuda/kernels/gptq_faster_fp16_bias.hpp"

void matmul(void* output, nQWeight_fp16 &nqW, void* input, int n, int algo);
void matmul(void* output, void* input, nQWeight_fp16 &nqW, int m, int algo);



/* float16 */
inline void matmul_useCublas(__half* output, nQWeight_fp16 &nqW, __half* input, int n);
inline void matmul_useCublas(__half* output, __half* input, nQWeight_fp16 &nqW, int m);
/************************** float16 ***********************/

void matmul_gptq(
    int m, int n, int k, void *scale, void *bias,
    void *A, void *B, void *C){
    hipMemset(C, 0, sizeof(__half) * m * n);
    kernel::gptq(n, k, (__half*)scale, (__half*)bias,
                (__half*)A, (uint32_t*)B, (__half*)C);
}

void matmul_gptq_faster(
    int m, int n, int k, void *scale, void *bias,
    void *A, void *B, void *C){
    hipMemset(C, 0, sizeof(__half) * m * n);
    kernel::gptq_faster(n, k, (__half*)scale, (__half*)bias,
                (half2*)A, (uint32_t*)B, (__half*)C);
}

void matmul(void* output, nQWeight_fp16 &nqW, void* input, int n, int algo){
    if(n==1){
        hipMemset(output, 0, sizeof(__half) * nqW.mSize);  // 0.007ms 0.04
        if(nqW.q_bias == nullptr)  kernel::nqmv((__half*)output, nqW, (__half*)input, algo);
        else                       kernel::nqmv_bias((__half*)output, nqW, (__half*)input, algo);
    } 
    else     matmul_useCublas((__half*)output, nqW, (__half*)input, n);
}
void matmul(void* output, void* input, nQWeight_fp16 &nqW, int m, int algo){
    if(m==1){
        hipMemset(output, 0, sizeof(__half) * nqW.mSize);
        if(nqW.q_bias == nullptr)  kernel::nqmv((__half*)output, nqW, (__half*)input, algo);
        else                       kernel::nqmv_bias((__half*)output, nqW, (__half*)input, algo);
    } 
    else     matmul_useCublas((__half*)output, (__half*)input, nqW, m);
}

inline void matmul_useCublas(__half* output, nQWeight_fp16 &nqW, __half* input, int n) {
    kernel::cublas_gemm_ex((__half*)nqW.getDequantiedWeight(true), input, output, nqW.mSize, n, nqW.kSize);
}

inline void matmul_useCublas(__half* output, __half* input, nQWeight_fp16 &nqW, int m) {
    kernel::cublas_gemm_ex(input, (__half*)nqW.getDequantiedWeight(true), output, m, nqW.mSize, nqW.kSize);
}

}



