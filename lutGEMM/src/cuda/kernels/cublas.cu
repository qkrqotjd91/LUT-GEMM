#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

namespace kernel{

template <typename T, typename S>
inline hipblasStatus_t cublas_gemm_ex(T *A,  T *B,  S *C,
                                    int m, int n, int k);

typedef hipblasStatus_t<__half

template <typename T, typename S>
inline hipblasStatus_t cublas_gemm_ex(T *A,  T *B,  S *C,
                                    int m, int n, int k) {
    static S alpha = 1;
    static S beta  = 0;
    static hipblasHandle_t handle = nullptr;
    if(handle == nullptr) hipblasCreate(&handle);
    
    hipDataType AType, BType, CType;
    hipblasComputeType_t  ComputeType;
    if (std::is_same<T, float>::value) {
        AType = BType = CType = HIP_R_32F;
        ComputeType = HIPBLAS_COMPUTE_32F_FAST_TF32;
    } else if (std::is_same<T, __half>::value) {
        AType = BType = CType = HIP_R_16F;
        ComputeType = HIPBLAS_COMPUTE_16F;
    } else if (std::is_same<T, int8_t>::value) {
        AType = BType = HIP_R_8I;
        CType = HIP_R_32I;
        ComputeType = HIPBLAS_COMPUTE_32I;
    } else {
        printf("Not supported data type.");
        return HIPBLAS_STATUS_NOT_SUPPORTED;
    }
    return hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                          n, m, k, 
                          &alpha,
                          B, BType, n,
                          A, AType, k,
                          &beta,
                          C, CType, n,
                          ComputeType,
                          HIPBLAS_GEMM_DEFAULT);
}

}
